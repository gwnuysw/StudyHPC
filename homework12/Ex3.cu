
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10000
#define THREAD_X 4

__global__ void index(float *A, float *B, float *C){
   int i = blockDim.x*blockIdx.x+threadIdx.x;

   C[i] = A[i] + B[i];
}
__global__ void inIt(float *A, float *B){
   int i = blockDim.x*blockIdx.x+threadIdx.x;

   A[i] = i*2;
   B[i] = N-i;
}

int main(){
   float A[N], *A_d;
   float B[N], *B_d;
   float C[N], *C_d;
   int i;


   dim3 dimBlock(THREAD_X);
   dim3 dimGrid(N/THREAD_X);

   hipMalloc((void**)&A_d, sizeof(int)*N);
   hipMalloc((void**)&B_d, sizeof(int)*N);
   hipMalloc((void**)&C_d, sizeof(int)*N);


   hipMemcpy(A_d, A, sizeof(int)*N, hipMemcpyHostToDevice);
   hipMemcpy(B_d, B, sizeof(int)*N, hipMemcpyHostToDevice);

   inIt<<<dimGrid, dimBlock>>>(A_d, B_d);

   hipMemcpy(C_d, C, sizeof(int)*N, hipMemcpyHostToDevice);

   index<<<dimGrid, dimBlock>>>(A_d, B_d, C_d);

   hipMemcpy(A, A_d, sizeof(int)*N, hipMemcpyDeviceToHost);
   hipMemcpy(B, B_d, sizeof(int)*N, hipMemcpyDeviceToHost);
   hipMemcpy(C, C_d, sizeof(int)*N, hipMemcpyDeviceToHost);

   for(i = 0; i < N; i++){
      printf("%f ",C[i]);
   }
   hipFree(B_d);
   hipFree(C_d);
   hipFree(A_d);
}
