
#include <hip/hip_runtime.h>
#include <stdio.h>

#define X 12

#define THREAD_X 4

__global__ void index(int *A){
   int i = blockDim.x*blockIdx.x+threadIdx.x;
   //int i = threadIdx.x;

   A[i] = i;
//   A[i] = gridDim.x;
   //A[i] = blockDim.x;
//   A[i] = threadIdx.x;
}

int main(){
   int A[X], *A_d;
   int i;

   //dim3 dimBlock(THREAD_X, THREAD_Y);
   //dim3 dimGrid(X/THREAD_X,Y/THREAD_Y);

   hipMalloc((void**)&A_d, sizeof(int)*X);

   for(i = 0; i < X; i++){
      A[i] = -1;
   }

   hipMemcpy(A_d, A, sizeof(int)*X, hipMemcpyHostToDevice);

   index<<<12/4, 4>>>(A_d);
   //index<<<dimGrid, dimBlock>>>(A_d);


   hipMemcpy(A, A_d, sizeof(int)*X, hipMemcpyDeviceToHost);

   for(i=0; i < X; i++){
      printf("%d ",A[i]);
   }
   printf("\n");
   hipFree(A_d);
}
