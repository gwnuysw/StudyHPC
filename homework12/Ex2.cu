
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000
#define THREAD_X 4

__global__ void index(float *A, float *B){
   int i = blockDim.x*blockIdx.x+threadIdx.x;
   //int i = threadIdx.x;
   float X = 1.23;
   float Y = 2.34 ;

   B[i] = A[i]*X + Y;
}

int main(){
   float A[N], *A_d;
   float B[N], *B_d;
   int i;

   dim3 dimBlock(THREAD_X);
   dim3 dimGrid(N/THREAD_X);

   for(i = 0 ; i < N; i++){
      A[i] = i*2;
   }

   hipMalloc((void**)&A_d, sizeof(int)*N);
   hipMalloc((void**)&B_d, sizeof(int)*N);

   hipMemcpy(A_d, A, sizeof(int)*N, hipMemcpyHostToDevice);
   hipMemcpy(B_d, B, sizeof(int)*N, hipMemcpyHostToDevice);

   index<<<dimGrid, dimBlock>>>(A_d, B_d);

   hipMemcpy(A, A_d, sizeof(int)*N, hipMemcpyDeviceToHost);
   hipMemcpy(B, B_d, sizeof(int)*N, hipMemcpyDeviceToHost);

   for(i = 0; i < N; i++){
      printf("%f ",B[i]);
   }

   hipFree(A_d);
}
