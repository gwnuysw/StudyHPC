
#include <hip/hip_runtime.h>
# include <stdio.h>
# include <stdlib.h>

#define M 8
#define N 9

#define THREAD_X 2
#define THREAD_Y 3

#define A(i,j) A[i*N+j]
#define B(i,j) B[i*N+j]
#define C(i,j) C[i*N+j]

__global__ void matAdd(int *A, int *B, int *C){
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   int j = blockDim.y * blockIdx.y + threadIdx.y;

   A(i,j) = B(i,j) + C(i,j);
}

int main(){
   int A[M][N], B[M][N], C[M][N];
   int *A_d, *B_d, *C_d;
   int i, j;

   dim3 dimBlock(THREAD_X, THREAD_Y);
   dim3 dimGrid(M/THREAD_X, N/THREAD_Y);

   hipMalloc((void**)&A_d, M*N*sizeof(int));
   hipMalloc((void**)&B_d, M*N*sizeof(int));
   hipMalloc((void**)&C_d, M*N*sizeof(int));

   for(i = 0; i < M; i++){
      for(j = 0; j < N;j++){
         B[i][j] = i*j;
         C[i][j] = i+j;
         //printf("%d ",B[i][j]);
      }
      //printf("\n");
   }

   hipMemcpy(A_d, A, sizeof(int)*M*N, hipMemcpyHostToDevice);
   hipMemcpy(B_d, B, sizeof(int)*M*N, hipMemcpyHostToDevice);
   hipMemcpy(C_d, C, sizeof(int)*M*N, hipMemcpyHostToDevice);

   matAdd<<<dimGrid, dimBlock>>>(A_d, B_d, C_d);

   hipMemcpy(A, A_d, sizeof(int)*M*N, hipMemcpyDeviceToHost);
   
   for(i = 0; i < M; i++){
      for(j = 0; j < N;j++){
         printf("%d ", A[i][j]);
      }
      printf("\n");
   }

   hipFree(A_d);
   hipFree(B_d);
   hipFree(C_d);
}
