
#include <hip/hip_runtime.h>
#include <stdio.h>

#define X 9
#define Y 8

#define THREAD_X 3
#define THREAD_Y 2

#define A(i, j) A[i*Y + j]

__global__ void index(int *A){
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   int j = blockDim.y * blockIdx.y + threadIdx.y;
   
   //A(i, j) = threadIdx.x;
   A(i, j) = threadIdx.y;
   //A(i ,j) = blockIdx.y;
}
int main(){
   int A[X][Y], *A_d;
   int i, j;

   dim3 dimBlock(THREAD_X, THREAD_Y);
   dim3 dimGrid(X/THREAD_X, Y/THREAD_Y);

   hipMalloc((void**)&A_d, sizeof(int)*X*Y);

   for(i = 0; i < X; i++)
      for(j = 0; j < Y; j++)
         A[i][j] = -1;

   hipMemcpy(A_d, A, sizeof(int)*X*Y, hipMemcpyHostToDevice);

   index<<<dimGrid, dimBlock>>>(A_d);

   hipMemcpy(A, A_d, sizeof(int)*X*Y, hipMemcpyDeviceToHost);

   for(i = 0; i < X; i++){
      for(j = 0; j < Y; j++){
         printf("%d ", A[i][j]);
      }
      printf("\n");
   }
   hipFree(A_d);
}
