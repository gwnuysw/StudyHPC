
#include <hip/hip_runtime.h>
# include <stdio.h>
# include <math.h>
# include <sys/time.h>

# define N 1000000
# define RADIUS 100
# define THREADS 32

__global__ void QuarterAreaOfCircle (  float *area , float *start, float *end){

   //int i = blockDim.x*blockIdx.x+threadIdx.x;
   int i = 0;
   float threadStartX;
   float x, dx;

   float segmentArea;

   // x starting value of each block
   
   threadStartX = ((float)RADIUS/(float)blockDim.x);
   start[threadIdx.x] = (float)threadIdx.x * threadStartX;
   x = start[threadIdx.x];
   end[threadIdx.x] = x;

   // increasing value of x
   dx = (float)RADIUS/(float)N;

   // calculate segment area
   for(i = 0; i < ((float)N/(float)blockDim.x); i++){
      x += dx;
      segmentArea += sqrt(fabs((float)RADIUS*(float)RADIUS-x*x)) * dx;
   }
   end[threadIdx.x] = dx;
   area[threadIdx.x] = segmentArea;
}

int main(int argc, char *argv[])
{
   float *reduceArea_d, reduceArea[THREADS], Area = 0;
   float *start_d, start[THREADS];
   float *end_d, end[THREADS];
   int i;

   dim3 dimBlock(THREADS);
   dim3 dimGrid(1);

   for( i = 0; i < dimBlock.x; i++){
      reduceArea[i] = 0;
   }

   hipMalloc( (void**) &reduceArea_d, sizeof(float) * THREADS );
   hipMalloc( (void**) &start_d, sizeof(float) * THREADS );
   hipMalloc( (void**) &end_d, sizeof(float) * THREADS );

   QuarterAreaOfCircle<<<dimGrid, dimBlock>>>(reduceArea_d, start_d, end_d);

   hipMemcpy(reduceArea, reduceArea_d, sizeof(float)*dimBlock.x, hipMemcpyDeviceToHost);
   hipMemcpy(start, start_d, sizeof(float)*dimBlock.x, hipMemcpyDeviceToHost);
   hipMemcpy(end, end_d, sizeof(float)*dimBlock.x, hipMemcpyDeviceToHost);

   for(i = 0; i < dimBlock.x; i++){
     Area += reduceArea[i]; 
     printf("reduced area : %5.10f , grid : %d,  area : %5.10f, sart : %5.10f, end : %5.10f \n", reduceArea[i], i, Area, start[i], end[i]);
   }

   printf("area : %5.10f \n",Area*4);
   hipFree(reduceArea_d);
   hipFree(start_d);
   hipFree(end_d);
}
