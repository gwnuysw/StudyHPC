
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 160
#define THREADS 16

__global__ void max_kernel(float *A, float * max){

   int i = blockDim.x*blockIdx.x+threadIdx.x;

   __shared__ float smax[THREADS];

   smax[threadIdx.x] = A[i];

   for(unsigned int s = blockDim.x/2; s > 0; s>>=1){
      if(threadIdx.x < s){
         if(smax[threadIdx.x] < smax[threadIdx.x+s]){
            smax[threadIdx.x] = smax[threadIdx.x+s];
            __syncthreads();
         }
      }
   }
   if(threadIdx.x == 0){
      *max = smax[threadIdx.x];
   }
}
int main(){
   float A[N], max, *maxs;
   float *A_d, *max_d;
   int i;

   dim3 dimBlock(THREADS);
   dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x);

   max = -10000.0;
   for(i = 0; i < N;i++){
      A[i] = rand()/(float)(1<<30);
      printf("%5.2f ", A[i]);
      if(A[i] > max)
         max = A[i];
   }
   printf("\nCPU -- %5.2f\n", max);

   hipMalloc((void**) &A_d, sizeof(float)*N);
   hipMalloc((void**) &max_d, sizeof(float)*dimGrid.x);

   hipMemcpy(A_d, A, sizeof(float)*N, hipMemcpyHostToDevice);

   max_kernel <<<dimGrid, dimBlock>>> (A_d, max_d);

   maxs = (float*)malloc(sizeof(float)*dimGrid.x);
   hipMemcpy(maxs, max_d, sizeof(float)*dimGrid.x, hipMemcpyDeviceToHost);

   max = -10000.0;
   for(i = 0;i<dimGrid.x;i++){
      if(maxs[i] > max){
         max = maxs[i];
      }
   }

   printf("CPU--%5.2f \n", max);

   hipFree(A_d);
   hipFree(max_d);
   free(maxs);
}
