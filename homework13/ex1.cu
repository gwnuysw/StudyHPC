
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16
#define THREADS 32

__global__ void max_kernel(float *A, float * max){
   __shared__ float smax[THREADS];

   smax[threadIdx.x] = A[threadIdx.x];

   for(unsigned int s = blockDim.x/2; s > 0; s>>=1){
      if(threadIdx.x < s){
         if(smax[threadIdx.x] < smax[threadIdx.x+s]){
            smax[threadIdx.x] = smax[threadIdx.x+s];
            __syncthreads();
         }
      }
   }
   if(threadIdx.x == 0){
      *max = smax[threadIdx.x];
   }
}
int main(){
   float A[N], max;
   float *A_d, *max_d;

   int i;
   dim3 dimBlock(THREADS);
   dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x);
   max = -10000.0;
   for(i = 0; i < N;i++){
      A[i] = rand()/(float)(1<<30);
      printf("%5.2f ", A[i]);
      if(A[i] > max)
         max = A[i];
   }
   printf("\nCPU -- %5.2f\n", max);

   hipMalloc((void**) &A_d, sizeof(float)*N);
   hipMalloc((void**) &max_d, sizeof(float));
   hipMemcpy(A_d, A, sizeof(float)*N, hipMemcpyHostToDevice);

   max_kernel <<<dimGrid, dimBlock>>> (A_d, max_d);

   max = -1.0;
   hipMemcpy(&max, max_d, sizeof(float), hipMemcpyDeviceToHost);

   printf("CPU--%5.2f \n", max);

   hipFree(A_d);
   hipFree(max_d);
}

