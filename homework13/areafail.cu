
#include <hip/hip_runtime.h>
# include <stdio.h>
# include <math.h>
# include <sys/time.h>

# define N 1000000
# define RADIUS 100
# define THREADS 32

__global__ void QuarterAreaOfCircle ( float *area ){

   //int i = blockDim.x*blockIdx.x+threadIdx.x;
   float blockStartX;
   float XofEachIdx, dx;

   __shared__ float segmentArea[THREADS];

   // x starting value of each block
   blockStartX = ((float)blockIdx.x * (float)(RADIUS/gridDim.x));

   // increasing value of x
   dx = (float)RADIUS/(float)N;

   // X value of each thread
   XofEachIdx = blockStartX + ((float)threadIdx.x * dx);

   // calculate segment area
   segmentArea[threadIdx.x] = sqrt(fabs((float)RADIUS*(float)RADIUS-XofEachIdx*XofEachIdx)) * dx;
   __syncthreads();
   // reduce 32 threads area to one
   for(unsigned int s = 0; s < threadIdx.x; s++){
      area[blockIdx.x] += segmentArea[s];
      __syncthreads();
   }
}

int main(int argc, char *argv[])
{
   float *reduceArea_d, reduceArea[N/THREADS], Area = 0;
   int i;

   dim3 dimBlock(THREADS);
   dim3 dimGrid(N/dimBlock.x);

   hipMalloc( (void**) &reduceArea_d, sizeof(float) * dimGrid.x );

   QuarterAreaOfCircle<<<dimGrid, dimBlock>>>(reduceArea_d);

   hipMemcpy(reduceArea, reduceArea_d, sizeof(float)*dimGrid.x, hipMemcpyDeviceToHost);

   for(i = 0; i < dimGrid.x; i++){
     Area += reduceArea[i]; 
     //printf("reduced area : %5.10f , grid : %d,  area : %5.10f\n", reduceArea[i], i, Area);
   }

   printf("area : %5.10f\n",Area*4);
   hipFree(reduceArea_d);
}
